﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <random>
#include "time.h"
#include <vector>

__global__ void MatrixMultiplicationGPU(const float* A, const float* B, float* C, size_t n)
{
	size_t C_index = blockIdx.x * blockDim.x + threadIdx.x;
	size_t A0_index = C_index/n;
	size_t B0_index = C_index - (A0_index*n);
	float sum = 0;
	for (auto temp = 0; temp < n; ++temp)
		sum += A[A0_index + temp] * B[B0_index + temp*n];
	C[C_index] = sum;
}

std::vector<std::vector<float>> MatrixMultiplicationCPU(const float* A,const float* B, size_t n)
{
	std::vector<std::vector<float>> C(n,std::vector<float>(n,0));
	float sum = 0;
	for (auto i = 0; i < n; ++i)
	{
		for (auto j = 0; j < n; ++j)
		{
			for (auto k = 0; k < n; ++k)
			{
				sum += A[i*n + k] * B[k*n + j];
			}
			C[i][j] = sum;
			sum = 0;
		}
	}
	return C;
}
int main()
{
	std::random_device rd;
	std::mt19937 gen(rd());

	size_t n = static_cast<size_t>(gen() % 1900) + 100;
	printf("Matrix dimentions are %dx%d\n", n, n);
	size_t matrixSize = n * n * sizeof(float);
	float *host_A = (float*)malloc(matrixSize);
	float *host_B = (float*)malloc(matrixSize);
	float *host_C = (float*)malloc(matrixSize);

	for (auto i = 0; i < n; ++i)
	{
		for (auto j = 0; j < n; ++j)
		{
			host_A[i*n + j] = static_cast<float>(gen());
			host_B[i*n + j] = static_cast<float>(gen());
		}
	}

	float* dev_A = nullptr;
	float* dev_B = nullptr;
	float* dev_C = nullptr;

	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_A, matrixSize);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_B, matrixSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_C, matrixSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
	}
	
	dim3 blocksPerGrid = dim3(((n*n) / 1024 + 1));
	dim3 threadsPerBlock = dim3(((n / 1024) ? 1024 : n));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	cudaStatus = hipMemcpy(dev_A, host_A, matrixSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy failed!");
	}
	cudaStatus = hipMemcpy(dev_B, host_B, matrixSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
	}

	MatrixMultiplicationGPU<<<blocksPerGrid,threadsPerBlock>>>(dev_A, dev_B, dev_C, n);

	cudaStatus = hipMemcpy(host_C, dev_C, matrixSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed! %d", cudaStatus);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float ElapsedTime;
	hipEventElapsedTime(&ElapsedTime, start, stop);
	printf("Time spent on GPU calculation: %.3f milliseconds\n",ElapsedTime);
	

	clock_t begin, end;
	begin = clock();
	//std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	auto vector = MatrixMultiplicationCPU(host_A, host_B, n);

	end = clock();
	//std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	float cputime = ((float)end - (float)begin)/ CLOCKS_PER_SEC;
	printf("Time spent on CPU calculation: %.3f seconds\n", cputime);

	free(host_A);
	free(host_B);
	free(host_C);
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	return 0;
}
/*
Device name : NVIDIA GeForce GTX 1050
Total global memory : 4095 MB
Shared memory per block : 49152
Registers per block : 65536
Warp size : 32
Memory pitch : 2147483647
Max threads per block : 1024
Max threads dimensions : x = 1024, y = 1024, z = 64
Max grid size: x = 2147483647, y = 65535, z = 65535
Clock rate: 1493000
Total constant memory: 65536
Compute capability: 6.1
Texture alignment: 512
Device overlap: 1
Multiprocessor count: 5
Kernel execution timeout enabled: true
*/