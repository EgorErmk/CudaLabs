﻿
#include "hip/hip_runtime.h"

#include <random>
#include <stdio.h>
#include <vector>
#include <chrono>

__global__ void VectorAdditionGPU(float *c, const float *a, const float *b)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

std::vector<float> VectorAdditionCPU(const float* A, const float* B, size_t n)
{
    std::vector<float> C((n*n),0);
    for (auto i = 0; i < (n*n); ++i)
    {
        C[i] = A[i] + B[i];
    }
    return C;
}

int main()
{
    std::random_device rd;
    std::mt19937 gen(rd());

    size_t n = static_cast<size_t>(gen() % 990) + 10;
    printf("Vector dimensions are 1x%d\n", (n*n));
    size_t matrixSize = n * n * sizeof(float);
    float* host_A = (float*)malloc(matrixSize);
    float* host_B = (float*)malloc(matrixSize);
    float* host_C = (float*)malloc(matrixSize);

    for (auto i = 0; i < n; ++i)
    {
        for (auto j = 0; j < n; ++j)
        {
            host_A[i * n + j] = static_cast<float>(gen());
            host_B[i * n + j] = static_cast<float>(gen());
        }
    }

    float* dev_A = nullptr;
    float* dev_B = nullptr;
    float* dev_C = nullptr;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_A, matrixSize);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_B, matrixSize);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_C, matrixSize);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }

    dim3 blocksPerGrid = dim3(((n * n) / 1024 + 1));
    dim3 threadsPerBlock = dim3(1024);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    cudaStatus = hipMemcpy(dev_A, host_A, matrixSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
    }
    cudaStatus = hipMemcpy(dev_B, host_B, matrixSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
    }

    VectorAdditionGPU<<<blocksPerGrid, threadsPerBlock>>> (dev_C, dev_A, dev_B);

    cudaStatus = hipMemcpy(host_C, dev_C, matrixSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed! %d", cudaStatus);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float ElapsedTime;
    hipEventElapsedTime(&ElapsedTime, start, stop);
    printf("Time spent on GPU calculation: %.3f milliseconds\n", ElapsedTime);

    auto begin = clock();
    
    auto vector = VectorAdditionCPU(host_A, host_B, n);

    float cputime = 1000*(float)(clock() - begin)/CLOCKS_PER_SEC;

    printf("Time spent on CPU calculation: %.3f milliseconds\n", cputime);

    printf("Precision test started...\n");
    int coarsecount = 0;
    for (auto i = 0; i < (n*n); ++i)
    {
        if ((host_C[i] - vector[i]) != 0)
        {
          coarsecount++;
        }
     }
    printf("Precision test finished. Ammount of coarse numbers: %d\n", coarsecount);

    free(host_A);
    free(host_B);
    free(host_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    return 0;
}
