#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/shuffle.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <random>
#include <cmath>
#include <iostream>
#include <thrust/sequence.h>

const unsigned int datapoints = 1000, population_size = 1000;
__constant__ float a = -4, b = 2, mean = 0, stddev = 1; // "a" is the left point "b" is the right
const float C0 = 2, C1 = 1, C2 = 1, C3 = -1, C4 = -0.5; // coefs 

typedef thrust::tuple<float, float, float, float, float> coefficients;


struct fitnesselement
{
	__host__ __device__
	float operator()(thrust::tuple<float&, float&>& temp, coefficients& coefs)
	{
		return	thrust::get<0>(coefs) + thrust::get<1>(coefs) * thrust::get<1>(temp) + thrust::get<2>(coefs) * thrust::get<1>(temp) * thrust::get<1>(temp) + thrust::get<3>(coefs) * thrust::get<1>(temp) * thrust::get<1>(temp) * thrust::get<1>(temp) + thrust::get<4>(coefs) * thrust::get<1>(temp) * thrust::get<1>(temp) * thrust::get<1>(temp) * thrust::get<1>(temp) - thrust::get<0>(temp);
	}
};
template<typename T>
struct absolute_value
{
	__host__ __device__ 
	T 	operator()(const T& x) const
	{
		return x < T(0) ? -x : x;
	}
};
struct crossover
{
	 __device__
	coefficients operator()(thrust::tuple<coefficients&, coefficients&>& temp, unsigned int seed)
	{
		coefficients child = thrust::get<1>(temp);
		thrust::get<1>(child) = thrust::get<1>(thrust::get<0>(temp));
		thrust::get<3>(child) = thrust::get<3>(thrust::get<0>(temp));
		return child;
	}
};
struct mutation
{
	__device__
	coefficients operator()(coefficients& temp, unsigned int seed)
	{
		hiprandState state;
		coefficients mutating_temp = temp;
		hiprand_init(seed, 0, 0, &state);
		uint32_t mask = 0, mask_ref = 0, cast = 0, value = 0;
		
		unsigned char k, n = static_cast<unsigned int>(hiprand_log_normal(&state, mean, stddev));
		for (size_t i = 0; i < n; i++)
		{
			k = static_cast<unsigned int>(hiprand_uniform(&state) * 10000) % 33;
			mask_ref = 0x80000000;
			mask_ref >>= k;
			mask |= mask_ref;
		}
		uint32_t* cast_ptr = reinterpret_cast<uint32_t*>(&thrust::get<0>(mutating_temp));
		cast = cast_ptr[0];
		value = ~cast & mask;
		cast &= ~mask;
		cast |= value;
		cast_ptr[0] = cast;
		cast_ptr = reinterpret_cast<uint32_t*>(&thrust::get<1>(mutating_temp));
		cast = cast_ptr[0];
		value = ~cast & mask;
		cast &= ~mask;
		cast |= value;
		cast_ptr[0] = cast;
		cast_ptr = reinterpret_cast<uint32_t*>(&thrust::get<2>(mutating_temp));
		cast = cast_ptr[0];
		value = ~cast & mask;
		cast &= ~mask;
		cast |= value;
		cast_ptr[0] = cast;
		cast_ptr = reinterpret_cast<uint32_t*>(&thrust::get<3>(mutating_temp));
		cast = cast_ptr[0];
		value = ~cast & mask;
		cast &= ~mask;
		cast |= value;
		cast_ptr[0] = cast;
		cast_ptr = reinterpret_cast<uint32_t*>(&thrust::get<4>(mutating_temp));
		cast = cast_ptr[0];
		value = ~cast & mask;
		cast &= ~mask;
		cast |= value;
		cast_ptr[0] = cast;
		return mutating_temp;
	}
};
//struct generate //for tests
//{
//	__device__
//	float operator()(float& seed)
//	{
//		float temp = seed;
//		uint32_t* cast_ptr = reinterpret_cast<uint32_t*>(&temp);
//		unsigned int mask = 0xc20, cast = 0, value = 0;
//		cast = cast_ptr[0];
//		value = ~cast & mask;
//		cast &= ~mask;
//		cast |= value;
//		cast_ptr[0] = cast;
//		return temp;
//	}
//};
int main()
{
	thrust::host_vector<float> H_initial_dataset(datapoints, 0);
	thrust::device_vector<float> D_initial_dataset(datapoints);
	thrust::device_vector<float> step(datapoints);
	thrust::host_vector<coefficients> H_population(population_size, (0, 0, 0, 0, 0));
	float h = 0;
	h = (b-a) / static_cast<float>(datapoints);
	std::random_device rd;
	std::mt19937 gen(rd());
	for (size_t i = 0; i < datapoints; i++)
	{
		H_initial_dataset[i] = C4 * pow((a + i * h), 4) + C3 * pow((a + i * h), 3) + C2 * pow((a + i * h), 2) + C1 * (a + i * h) + C0 + (static_cast<float>(gen()%2000)/1000 - 1); // y = C₄x⁴ - C₃x³ + C₂x² + C₁x¹ + C₀ + rnd(0,1)
		step[i] = (a + i * h);
	}
	
	D_initial_dataset = H_initial_dataset;

	//fitness vars

	thrust::device_vector<coefficients> D_population(population_size, (0,0,0,0,0));
	thrust::device_vector<float> deviation(datapoints);
	thrust::device_vector<float> fitness(datapoints);
	thrust::device_vector<coefficients> parents(population_size / 2);

	//crossover vars

	thrust::device_vector<coefficients> shuffled_parents(population_size / 2);
	thrust::device_vector<coefficients> children(population_size / 2);
	thrust::default_random_engine g;
	unsigned int generations = 1;
	std::cin >> generations;

	for (size_t j = 0; j < generations; j++)
	{
		// fitness calculation

		for (size_t i = 0; i < population_size; i++)
		{
			thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(D_initial_dataset.begin(), step.begin())), thrust::make_zip_iterator(thrust::make_tuple(D_initial_dataset.end(), step.end())), thrust::make_constant_iterator(D_population[i]), deviation.begin(), fitnesselement());
			fitness[i] = thrust::transform_reduce(deviation.begin(), deviation.end(), absolute_value<float>(), 0, thrust::maximum<float>());
		}

		// selection

		thrust::sort_by_key(fitness.begin(), fitness.end(), D_population.begin());
		thrust::copy_n(thrust::device, D_population.begin(), population_size / 2, parents.begin());

		//crossover generation

		thrust::shuffle_copy(thrust::device, parents.begin(), parents.end(), shuffled_parents.begin(), g);
		thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(parents.begin(), shuffled_parents.begin())), thrust::make_zip_iterator(thrust::make_tuple(parents.end(), shuffled_parents.end())), thrust::counting_iterator<int>(0), children.begin(), crossover());

		//mutation

		thrust::copy_n(thrust::device, parents.begin(), population_size / 2, D_population.begin());
		thrust::copy_n(thrust::device, children.begin(), population_size / 2, D_population.begin() + population_size / 2);

		thrust::transform(D_population.begin() + 1, D_population.end(), thrust::counting_iterator<int>(0), D_population.begin() + 1, mutation());
		thrust::copy_n(thrust::device, parents.begin(), 1, D_population.begin());

		//test
		//thrust::device_vector<float> test(population_size, 1.0);
		////thrust::sequence(thrust::device, test.begin(), test.end());
		//thrust::host_vector<float> host_test(population_size);
		//thrust::transform(test.begin(),test.end(), test.begin(), generate());
		//host_test = test;
		//for (size_t i = 0; i < 10; i++)
		//{
		//	std::cout << host_test[i] << '\n';
		//}
	}
	H_population = D_population;
	std::cout << thrust::get<0>(H_population[0]) << '\n';
	std::cout << thrust::get<1>(H_population[0]) << '\n';
	std::cout << thrust::get<2>(H_population[0]) << '\n';
	std::cout << thrust::get<3>(H_population[0]) << '\n';
	std::cout << thrust::get<4>(H_population[0]) << '\n';
		
}
